#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <fstream>
#include <vector>
#include "json.hpp" // JSONライブラリを使用

#define debug(var)                  \
    do                              \
    {                               \
        std::cout << #var << " : "; \
        view(var);                  \
    } while (0)
template <typename T>
void view(T e) { std::cout << e << std::endl; }
template <typename T>
void view(const std::vector<T> &v)
{
    for (const auto &e : v)
    {
        std::cout << e << " ";
    }
    std::cout << std::endl;
}
template <typename T>
void view(const std::vector<std::vector<T>> &vv)
{
    for (const auto &v : vv)
    {
        view(v);
    }
}

#define four9ths (4.0 / 9.0)
#define one9th (1.0 / 9.0)
#define one36th (1.0 / 36.0)

__global__ void stream(double *nN, double *nS, double *nE, double *nW, double *nNE, double *nNW, double *nSE, double *nSW, bool *barrier, int height, int width)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= height * width)
        return;

    int x = idx % width;
    int y = idx / width;

    // Streaming step
    nN[idx] = nN[(y - 1 + height) % height * width + x];
    nS[idx] = nS[(y + 1) % height * width + x];
    nE[idx] = nE[y * width + (x + 1) % width];
    nW[idx] = nW[y * width + (x - 1 + width) % width];
    nNE[idx] = nNE[((y - 1 + height) % height) * width + (x + 1) % width];
    nNW[idx] = nNW[((y - 1 + height) % height) * width + (x - 1 + width) % width];
    nSE[idx] = nSE[((y + 1) % height) * width + (x + 1) % width];
    nSW[idx] = nSW[((y + 1) % height) * width + (x - 1 + width) % width];

    // Bounce-back for barriers
    if (barrier[idx])
    {
        nN[idx] = nS[idx];
        nS[idx] = nN[idx];
        nE[idx] = nW[idx];
        nW[idx] = nE[idx];
        nNE[idx] = nSW[idx];
        nNW[idx] = nSE[idx];
        nSE[idx] = nNW[idx];
        nSW[idx] = nNE[idx];
    }
}

__global__ void collide(double *n0, double *nN, double *nS, double *nE, double *nW, double *nNE, double *nNW, double *nSE, double *nSW, double *rho, double *ux, double *uy, int height, int width, double omega)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= height * width)
        return;

    int i = idx / width;
    int j = idx % width;

    // Compute macroscopic quantities
    rho[idx] = n0[idx] + nN[idx] + nS[idx] + nE[idx] + nW[idx] + nNE[idx] + nSE[idx] + nNW[idx] + nSW[idx];
    ux[idx] = (nE[idx] + nNE[idx] + nSE[idx] - nW[idx] - nNW[idx] - nSW[idx]) / rho[idx];
    uy[idx] = (nN[idx] + nNE[idx] + nNW[idx] - nS[idx] - nSE[idx] - nSW[idx]) / rho[idx];

    double ux2 = ux[idx] * ux[idx];
    double uy2 = uy[idx] * uy[idx];
    double u2 = ux2 + uy2;
    double omu215 = 1 - 1.5 * u2;
    double uxuy = ux[idx] * uy[idx];
    double u0 = 0.1;

    // Collision step
    n0[idx] = (1 - omega) * n0[idx] + omega * four9ths * rho[idx] * omu215;
    nN[idx] = (1 - omega) * nN[idx] + omega * one9th * rho[idx] * (omu215 + 3 * uy[idx] + 4.5 * uy2);
    nS[idx] = (1 - omega) * nS[idx] + omega * one9th * rho[idx] * (omu215 - 3 * uy[idx] + 4.5 * uy2);
    nE[idx] = (1 - omega) * nE[idx] + omega * one9th * rho[idx] * (omu215 + 3 * ux[idx] + 4.5 * ux2);
    nW[idx] = (1 - omega) * nW[idx] + omega * one9th * rho[idx] * (omu215 - 3 * ux[idx] + 4.5 * ux2);
    nNE[idx] = (1 - omega) * nNE[idx] + omega * one36th * rho[idx] * (omu215 + 3 * (ux[idx] + uy[idx]) + 4.5 * (u2 + 2 * uxuy));
    nNW[idx] = (1 - omega) * nNW[idx] + omega * one36th * rho[idx] * (omu215 + 3 * (-ux[idx] + uy[idx]) + 4.5 * (u2 - 2 * uxuy));
    nSE[idx] = (1 - omega) * nSE[idx] + omega * one36th * rho[idx] * (omu215 + 3 * (ux[idx] - uy[idx]) + 4.5 * (u2 - 2 * uxuy));
    nSW[idx] = (1 - omega) * nSW[idx] + omega * one36th * rho[idx] * (omu215 + 3 * (-ux[idx] - uy[idx]) + 4.5 * (u2 + 2 * uxuy));

    // Boundary conditions for forced flow
    if (j == 0)
    { // Top boundary
        nE[idx] = (1.0 / 9.0) * (1 + 3 * u0 + 4.5 * u0 * u0 - 1.5 * u0 * u0);
        nNE[idx] = (1.0 / 36.0) * (1 + 3 * u0 + 4.5 * u0 * u0 - 1.5 * u0 * u0);
        nSE[idx] = (1.0 / 36.0) * (1 + 3 * u0 + 4.5 * u0 * u0 - 1.5 * u0 * u0);
    }
    else if (j == width - 1)
    { // Bottom boundary
        nW[idx] = (1.0 / 9.0) * (1 - 3 * u0 + 4.5 * u0 * u0 - 1.5 * u0 * u0);
        nNW[idx] = (1.0 / 36.0) * (1 - 3 * u0 + 4.5 * u0 * u0 - 1.5 * u0 * u0);
        nSW[idx] = (1.0 / 36.0) * (1 - 3 * u0 + 4.5 * u0 * u0 - 1.5 * u0 * u0);
    }


}

void load_config(const std::string &filename, int &height, int &width, double &viscosity, double &u0, int &total_steps, int &skip_frames, std::vector<std::tuple<int, int, int>> &barriers)
{
    std::ifstream file(filename);
    if (!file.is_open())
    {
        throw std::runtime_error("Cannot open config file");
    }

    // JSONファイルの内容を解析
    nlohmann::json config;
    file >> config;

    // 必要なパラメータをロード
    height = config["height"];
    width = config["width"];
    viscosity = config["viscosity"];
    u0 = config["u0"];
    total_steps = config["total_steps"];
    skip_frames = config["skip_frames"];


    // バリア情報を読み込む
    for (const auto &barrier : config["barrier"]) // 修正点: config["barriers"] → config["barrier"]
    {
        int start_row = barrier["start_row"]; // 修正点: barrier[0]["start_row"] → barrier["start_row"]
        int end_row = barrier["end_row"];     // 修正点: barrier[0]["end_row"] → barrier["end_row"]
        int col = barrier["col"];             // 修正点: barrier[0]["col"] → barrier["col"]
        barriers.emplace_back(start_row, end_row, col);
    }
}

// グリッドの初期化とバリア設定
void initialize_barrier(bool *barrier, std::vector<std::tuple<int, int, int>> &barriers, int height, int width)
{
    // バリアを設定
    for (int k = 0; k < barriers.size(); k++)
    {
        int start_row = std::get<0>(barriers[k]);
        int end_row = std::get<1>(barriers[k]);
        int col = std::get<2>(barriers[k]);

        for (int i = start_row; i < end_row; i++)
        {
            barrier[i * width + col] = true;
        }
    }
}

void save_to_json(const std::vector<std::vector<double>> &data, const std::string &filename, int height, int width)
{
    using json = nlohmann::json;
    json result;

    // フレームごとにデータをJSONに格納
    for (size_t frame = 0; frame < data.size(); frame++)
    {
        for (size_t i = 0; i < height; i++)
        {
            for (size_t j = 0; j < width; j++)
            {
                int idx = i * width + j;
                result[frame][i][j] = data[frame][idx];
            }
        }
    }

    // JSONファイルに書き込み
    std::ofstream file(filename);
    file << result.dump(4); // インデントを付けて書き込む
    file.close();
}

int main()
{
    // 計測開始
    clock_t start = clock();
    
    // 変数宣言
    int height, width, total_steps, skip_frames;
    double viscosity, u0;
    std::vector<std::tuple<int, int, int>> barriers;
    // 設定ファイルの読み込み
    load_config("setting.json", height, width, viscosity, u0, total_steps, skip_frames, barriers);
    // omegaの計算
    const double omega = 1.0 / (3.0 * viscosity + 0.5);
    std::cout << "omega:" << omega << std::endl;

    // Initialize arrays
    double *n0, *nN, *nS, *nE, *nW, *nNE, *nNW, *nSE, *nSW, *rho, *ux, *uy;
    bool *barrier;
    hipMallocManaged(&n0, height * width * sizeof(double));
    hipMallocManaged(&nN, height * width * sizeof(double));
    hipMallocManaged(&nS, height * width * sizeof(double));
    hipMallocManaged(&nE, height * width * sizeof(double));
    hipMallocManaged(&nW, height * width * sizeof(double));
    hipMallocManaged(&nNE, height * width * sizeof(double));
    hipMallocManaged(&nNW, height * width * sizeof(double));
    hipMallocManaged(&nSE, height * width * sizeof(double));
    hipMallocManaged(&nSW, height * width * sizeof(double));
    hipMallocManaged(&rho, height * width * sizeof(double));
    hipMallocManaged(&ux, height * width * sizeof(double));
    hipMallocManaged(&uy, height * width * sizeof(double));
    hipMallocManaged(&barrier, height * width * sizeof(bool));

    // Initialize particle densities and barriers
    for (int y = 0; y < height; ++y)
    {
        for (int x = 0; x < width; ++x)
        {
            int idx = y * width + x;
            n0[idx] = four9ths * (1 - 1.5 * u0 * u0);
            nN[idx] = one9th * (1 - 1.5 * u0 * u0);
            nS[idx] = one9th * (1 - 1.5 * u0 * u0);
            nE[idx] = one9th * (1 + 3 * u0 + 4.5 * u0 * u0 - 1.5 * u0 * u0);
            nW[idx] = one9th * (1 - 3 * u0 + 4.5 * u0 * u0 - 1.5 * u0 * u0);
            nNE[idx] = one36th * (1 + 3 * u0 + 4.5 * u0 * u0 - 1.5 * u0 * u0);
            nSE[idx] = one36th * (1 + 3 * u0 + 4.5 * u0 * u0 - 1.5 * u0 * u0);
            nNW[idx] = one36th * (1 - 3 * u0 + 4.5 * u0 * u0 - 1.5 * u0 * u0);
            nSW[idx] = one36th * (1 - 3 * u0 + 4.5 * u0 * u0 - 1.5 * u0 * u0);
            barrier[idx] = false;
        }
    }

    // Set barrier
    view("バリア設定スタート");
    initialize_barrier(barrier, barriers, height, width);

    // Main simulation loop
    std::vector<std::vector<double>> results;
    for (int step = 0; step < total_steps; ++step)
    {
        stream<<<(height * width + 255) / 256, 256>>>(nN, nS, nE, nW, nNE, nNW, nSE, nSW, barrier, height, width);
        collide<<<(height * width + 255) / 256, 256>>>(n0, nN, nS, nE, nW, nNE, nNW, nSE, nSW, rho, ux, uy, height, width, omega);
        hipDeviceSynchronize();

        if (step % skip_frames == 0)
        {
            std::vector<double> frame(height * width);
            for (int y = 0; y < height; ++y)
            {
                for (int x = 0; x < width; ++x)
                {
                    frame[y * width + x] = rho[y * width + x];
                }
            }
            results.push_back(frame);
        }
    }

    // Save results to JSON
    save_to_json(results, "result.json", height, width);

    // Free memory
    hipFree(n0);
    hipFree(nN);
    hipFree(nS);
    hipFree(nE);
    hipFree(nW);
    hipFree(nNE);
    hipFree(nNW);
    hipFree(nSE);
    hipFree(nSW);
    hipFree(rho);
    hipFree(ux);
    hipFree(uy);
    hipFree(barrier);

    return 0;
}